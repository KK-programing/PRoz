
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

string INPUT = "10x500.txt";
float MUTATION_RATE = 0.01;
int POPULATION_SIZE = 1000;
int TIME_LIMIT = 30;

int MUTATIONS_NUM;
int PROCS;
int PROCCESORS;
int CURRENT;
int MINN;

class GA
{
public:
    vector<int> ORDER;
    int LAST_MAX;
    int CURR;
    int CURR_SUM;
};

__global__ void fitness(int *tab, int max, int *fit_proc, int *fitnesReturn, int size)
{
    int x, s = 0, fp = *fit_proc;

    int row = threadIdx.x;
    if (row < 1000)
    {
        for (x = 0; x < size; x++)
        {
            if (s + tab[row * size + x] <= max)
                s += tab[row * size + x];
            else
            {
                s = tab[row * size + x];
                fp--;
                if (fp <= 0)
                    break;
            }
        }
    }
    int help = size - x;
    fitnesReturn[row] = help;
}

int fitnessHelper(vector<int> &tab, int r)
{
    int s = 0;
    for (int x = tab.size() - r; x < tab.size(); x++)
        s += tab[x];
    return s;
}

void shuffle(vector<int> &tab)
{
    for (int i = 0; i < tab.size(); ++i)
    {
        int j = rand() % tab.size();
        swap(tab[i], tab[j]);
    }
}

void doMutate(vector<int> &tab)
{
    for (int i = 0; i < MUTATIONS_NUM; i++)
    {
        int a = rand() % tab.size();
        int b = rand() % tab.size();
        swap(tab[a], tab[b]);
    }
}

int doFitnessAll(vector<GA> &pop, int max)
{
    hipError_t err;
    int *cuda_fitnessReturn;
    int fitnesReturn[POPULATION_SIZE];
    int *cuda_tab;
    int *cuda_PROCESORS;
    int size = pop[0].ORDER.size();
    int tab[POPULATION_SIZE * size];
    int cnt = 0;
    for (auto &g : pop)
    {
        g.LAST_MAX = max;
        copy(g.ORDER.begin(), g.ORDER.end(), tab + cnt * size);
        cnt++;
    }
    err = hipMalloc((int **)&cuda_fitnessReturn, sizeof(int) * POPULATION_SIZE);
    if (err != hipSuccess)
    {
        printf("CUDA: %s", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    err = hipMalloc((int **)&cuda_PROCESORS, sizeof(int));
    if (err != hipSuccess)
    {
        printf("CUDA: %s", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    err = hipMalloc((int **)&cuda_tab, size * POPULATION_SIZE * sizeof(int));
    hipMemcpy(cuda_tab, tab, size * POPULATION_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_PROCESORS, &PROCCESORS, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA: %s", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    fitness<<<1, POPULATION_SIZE>>>(cuda_tab, max, cuda_PROCESORS, cuda_fitnessReturn, size);
    hipMemcpy(fitnesReturn, cuda_fitnessReturn, sizeof(int) * POPULATION_SIZE, hipMemcpyDeviceToHost);
    hipFree(cuda_fitnessReturn);
    hipFree(cuda_PROCESORS);
    hipFree(cuda_tab);
    cnt = 0;
    for (auto &g : pop)
    {
        g.CURR = fitnesReturn[cnt];
        g.CURR_SUM = fitnessHelper(g.ORDER, g.CURR);
        cnt++;
    }
    return 0;
}

int fitnessCommon(vector<int> &tab, int max, int fit_proc)
{
    int x, s = 0;
    for (x = 0; x < tab.size(); x++)
    {
        if (s + tab[x] <= max)
            s += tab[x];
        else
        {
            s = tab[x];
            fit_proc--;
            if (fit_proc <= 0)
                break;
        }
    }
    return tab.size() - x;
}

int doFitness(GA *g, int max)
{
    if (g->LAST_MAX != max)
    {
        g->LAST_MAX = max;
        g->CURR = fitnessCommon(g->ORDER, max, PROCCESORS);
        g->CURR_SUM = fitnessHelper(g->ORDER, g->CURR);
    }
    return g->CURR;
}

void mutate(GA &g)
{
    g.LAST_MAX = 0;
    doMutate(g.ORDER);
}

bool compare(GA &a, GA &b)
{
    return a.CURR < b.CURR || (a.CURR == b.CURR && a.CURR_SUM < b.CURR_SUM);
}

int findBest(GA &a, int r)
{
    int l = MINN - 1;
    int m = (l + r) / 2;
    while (m > l)
    {
        doFitness(&a, m) == 0 ? r = m : l = m;
        m = (l + r) / 2;
    }
    return r;
}

int runBest(GA &a)
{
    int max = MINN * 2;
    while (doFitness(&a, max) != 0)
        max *= 2;
    return findBest(a, max);
}

vector<GA> nextGen(vector<GA> &pop)
{
    vector<GA> next;
    doFitnessAll(pop, CURRENT); // wykorzystanie CUDY
    sort(pop.begin(), pop.end(), compare);
    next.push_back(pop[0]);
    while (next.size() < pop.size()) // mutacja osobników
    {
        GA g = pop[rand() % (rand() % pop.size() + 1)];
        mutate(g);
        next.push_back(g);
    }
    if (next[0].CURR == 0)
        CURRENT = findBest(next[0], CURRENT) - 1;
    return next;
}

vector<int> getData()
{
    vector<int> tab;
    int temp;
    fstream file;
    file.open(INPUT, fstream::in);
    if (!file.good())
        cerr << "File error";
    file >> PROCCESORS >> PROCS;
    while (file >> temp)
        tab.push_back(temp), MINN += temp;
    MINN = MINN / PROCCESORS - MINN % PROCCESORS;
    return tab;
}

int main()
{
    int x = 9999999999;
    srand(time(nullptr));
    auto tab = getData();                              // pobranie danych z pliku
    MUTATIONS_NUM = (int)(MUTATION_RATE * tab.size()); // ustalenie liczby mutaacji na osobnika
    vector<GA> POPULATION;
    int iterations = 0;
    for (int x = 0; x < POPULATION_SIZE; x++) // tworzenie początkowej populacji
    {
        GA g;
        g.LAST_MAX = 0;
        g.ORDER = tab;
        shuffle(g.ORDER);
        POPULATION.push_back(g);
    }
    CURRENT = runBest(POPULATION[0]);
    while ((clock() / CLOCKS_PER_SEC) < TIME_LIMIT) // pilnowanie czasu
    {
        POPULATION = nextGen(POPULATION); // tworzenie nowej generacji
        if (x > CURRENT)
        {
            // cout << clock() / CLOCKS_PER_SEC << "--" << CURRENT << endl;
            x = CURRENT;
        }
        iterations++; // podbicie iteracji
    }
    cout << "Wynik koncowy: " << CURRENT << endl;
    cout << "Liczba iteracji: " << iterations << endl;
    return 0;
}